#include <stdio.h>
#include <fcntl.h>

// #include <hip/hip_runtime.h>
// #include <cufile.h>

#include "moduli-set.h"
#include "residue-int.h"

__host__
void ri_test()
{
    printf("ri_test here!\n");
}


// int ri_res_path(uint32_t i, char *path, struct ri *r)
// { return sprintf(path, "%s/%u.32b", r->dir, r->mset->moduli[i]) >= 0 ? 0 : -1; }

// int ri_init(char *dir, struct ms *mset, uint32_t digit_count, struct ri *r)
// {
//     // Set Structure Variables
//     r->dir = dir;
//     r->mset = mset;
//     r->nd = digit_count;
//     r->nm = mset->n;
//     r->ds = 4*r->nm; 

//     // Create files and allocate enough space
//     char path[RI_MAX_PATH_SIZE];
//     for(uint32_t i = 0; i < r->nm; ++i)
//     {
//         ri_res_path(i, path, r);
//         int fd = open(path, O_RDWR | O_CREAT, 0666);
//         if(fd < 0) return -1;
        
//         if(posix_fallocate(fd, 0, r->ds) != 0)
//             return -1;
        
//         close(fd);
//     }

//     return 0;
// }

// int ri_set_digit(uint32_t i, uint32_t k, struct ri *r)
// {
//     char path[RI_MAX_PATH_SIZE];
//     uint32_t n;
//     for(uint32_t j = 0; j < r->nm; ++j)
//     {
//         ri_res_path(j, path, r);
        
//         int fd = open(path, O_WRONLY);
//         if(fd < 0) return -1;

//         n = k%r->mset->moduli[j];
//         if(pwrite(fd, &n, 4, 4*i) == -1)
//             return -1;

//         close(fd);
//     }

//     return 0;
// }

// int ri_read_digit(uint32_t i, uint32_t *buf, struct ri *r)
// {
//     char path[RI_MAX_PATH_SIZE];
//     for(uint32_t j = 0; j < r->nm; ++j)
//     {
//         ri_res_path(j, path, r);
        
//         int fd = open(path, O_RDONLY);
//         if(fd == -1) return -1;

//         if(pread(fd, buf+j, 4, 4*i) == -1)
//             return -1;

//         close(fd);
//     }

//     return 0;
// }

// int ri_to_mrns(uint32_t fi, uint32_t si, struct ri *r)
// {
//     CUfileDescr_t cfd[r->nm];
//     CUfileHandle_t cfh[r->nm];
// }

// int ri_cufile_dig_setup(uint32_t n, CUfileDescr_t **cfd, CUfileDescr_t **cfd)
// {
    
// }
// {
//    memset((void *)cfd, 0, MAX_BATCH_IOS * sizeof(CUfileDescr_t));
// }

// int ri_cufile_dig_set(uint32_t fi, uint32_t si)
// {
    
// }

// int ri_cufile_dig_quit()
// {

// }

// int fds[r.nm]; // Posix file descriptors
// CUfileDescr_t cfd[r.nm]; // cuFile Descriptors
// CUfileHandle_t cfh[r.nm]; // cuFile handeler
// CUfileIOParams_t batch_parms[r.nm];
// CUfileBatchHandle_t batch_id;
// uint32_t *dev_buf;

// memset((void *)cfd, 0, r.nm*sizeof(CUfileDescr_t));
// for(uint32_t i = 0; i < r.nm; i++) {
//     fds[i] = open(residue_path(i, r).c_str(), O_RDWR | O_DIRECT);
//     posixCheck(fds[i]);
//     cfd[i].handle.fd = fds[i];
//     cfd[i].type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
//     cuFileCheck(cuFileHandleRegister(&cfh[i], &cfd[i]));
// }

// // Setup device buffer
// cudaCheck(hipMalloc(&dev_buf, r.nm*(fi - si)));
// cuFileCheck(cuFileBufRegister(dev_buf, r.nm*(fi - si), 0));

// // Setup batch IO
// for(uint32_t i = 0; i < r.nm; i++) {
//     batch_parms[i].mode = CUFILE_BATCH;
//     batch_parms[i].fh = cfh[i];
//     batch_parms[i].u.batch.devPtr_base = dev_buf;
//     batch_parms[i].u.batch.devPtr_offset = fi-si;
//     batch_parms[i].u.batch.file_offset = 4*si;
//     batch_parms[i].u.batch.size = 4*fi;
//     batch_parms[i].opcode = CUFILE_READ;
// }
// cuFileCheck(cuFileBatchIOSetUp(&batch_id, r.nm));