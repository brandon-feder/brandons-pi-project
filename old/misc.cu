#ifndef MISC_CU
#define MISC_CU

#include <hip/hip_runtime.h>

#include <iostream>
#include <string>
#include <sstream>
#include <cufile.h>

using namespace std;

// template<class T, typename enable_if<is_integral<T>::value, nullptr_t>::type = nullptr>
// std::string cuFileGetErrorString(T status) {
//    status = std::abs(status);
//    return IS_CUFILE_ERR(status) ?
//             std::string(CUFILE_ERRSTR(status)) : std::string(std::strerror(status));
// }

#define posixCheck(ans) { posixAssert((ans), __FILE__, __LINE__); }
#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
#define cuFileCheck(ans) { cuFileAssert((ans), __FILE__, __LINE__); }

inline void posixAssert(int code, const char *file, int line, bool abort=true)
{
   if(code < 0) 
   {
      stringstream err;
      err << strerror(errno) << " ";
      err << file << " ";
      err << line;

      throw runtime_error(err.str());
   }
}

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      stringstream err;
      err << hipGetErrorString(code) << " ";
      err << file << " ";
      err << line;

      throw runtime_error(err.str());
   }
}

inline void cuFileAssert(CUfileError_t code, const char *file, int line, bool abort=true)
{
   if (code.err != CU_FILE_SUCCESS) 
   {
      stringstream err;
      err << CUFILE_ERRSTR(code.err) << " ";
      err << file << " ";
      err << line;

      throw runtime_error(err.str());
   }
}


#endif